#include "hip/hip_runtime.h"
#include "boid.h"

#include <algorithm>
#include <random>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>
#include <fstream>
#include <cmath>

#include "vec3.h"

vec3* boid::pos = nullptr;
vec3* boid::vel = nullptr;
vec3* boid::acc = nullptr;
vec3* boid::sim_boids = nullptr;
vec3 boid::dim_low = vec3(-10., -10., -10.);
vec3 boid::dim_high = vec3(10., 10., 10.);
vec3 boid::vel_low = vec3(-1., -1., -1.);
vec3 boid::vel_high = vec3(1., 1., 1.);
vec3 boid::center = vec3(0., 0., 0.);
int boid::nboids = 2;
int boid::steps = 0;
float boid::dt = 1. / (float)(60);
float boid::time = 0.0;
float boid::centering_distance = 1.5;
float boid::alignment_distance = 1.;

float gtfo_distance = 1;
float boid::w_collision = 0.4;
float boid::w_alignment = 0.4;
float boid::w_centering = 0.3;

void boid::new_boids_random(){
    kill();
    pos = new vec3[nboids];
    vel = new vec3[nboids];
    acc = new vec3[nboids];
    vec3 dim_diff = dim_high - dim_low;
    vec3 vel_diff = vel_high - vel_low;
    for (int i = 0; i < 3*nboids; i++){
        ((float*)pos)[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        ((float*)pos)[i] *= ((float*)&dim_diff)[i%3];
        ((float*)pos)[i] += ((float*)&dim_low)[i%3];
        ((float*)vel)[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        ((float*)vel)[i] *= ((float*)&vel_diff)[i%3];
        ((float*)vel)[i] += ((float*)&vel_low)[i%3];
        ((float*)acc)[i] = 0;
    }
    for (int i = 0; i < nboids; i++) vel[i].normalize();
}

void boid::kill(){
    delete [] pos;
    delete [] vel;
    delete [] acc;
    delete [] sim_boids;
}

void boid::step_sim(){
    calc_acc_all();
    physics_update();
    time += dt;
}

__global__
void sim_kernel(int nboids, int steps, float time, float dt, float gtfo_distance,
                float centering_distance, float alignment_distance,
                float w_col, float w_ali, float w_cen,
                float* dpos, float *dvel, float *dacc,
                float* ddim_low, float* ddim_high,
                float* dvel_low, float* dvelhigh,
                float* data_array){
//stuff
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    //FIRST DATA_ARRAY ENTRY
    if (index < nboids){
        data_array[3*index] = dpos[3*index];
        data_array[3*index+1] = dpos[3*index+1];
        data_array[3*index+2] = dpos[3*index+2];
        // printf("%i, %i, %f, %f, %f \n",
        //             index,
        //             0,
        //             data_array[0*nboids*3 + 3*index],
        //             data_array[0*nboids*3 + 3*index+1],
        //             data_array[0*nboids*3 + 3*index+2]);
    }
    __syncthreads();

    float x, y, z, vx, vy, vz, acx, acy, acz;
    for(int time_step = 1; time_step < steps; time_step++){
        if (index < nboids){
                        
        //calc acc
            float cox=0, coy=0, coz=0;
            float alx=0, aly=0, alz=0;
            float cex=0, cey=0, cez=0;
            // vec3 collision = vec3(0,0,0);
            // vec3 alignment = vec3(0,0,0);
            // vec3 centering = vec3(0,0,0);
            float wax=0, way=0, waz=0;
            // vec3 wa_high = vec3(0,0,0);
            // vec3 wa_low = vec3(0,0,0);
            int collision_counter = 0;
            int alignment_counter = 1;
            int centering_counter = 1;
            x = dpos[3*index];
            y = dpos[3*index+1];
            z = dpos[3*index+2];
            vx = dvel[3*index];
            vy = dvel[3*index+1];
            vz = dvel[3*index+2];
            float avg_diffx=0, avg_diffy=0, avg_diffz=0;
            float avg_velx=vx, avg_vely=vy, avg_velz=vz;
            float avg_posx=x, avg_posy=y, avg_posz=z;
            // vec3 avg_diff = vec3(0,0,0);
            // vec3 avg_vel = vec3(vx,vy,vz);
            // vec3 avg_pos = vec3(x,y,z);

            //all in 1
            for (int j = 1; j < nboids; j++){
                int i = (j + index) % nboids;
                float jx = dpos[3*i];
                float jy = dpos[3*i+1];
                float jz = dpos[3*i+2];
                float dx = jx - x;
                float dy = jy - y;
                float dz = jz - z;
                float vjx = dvel[3*i];
                float vjy = dvel[3*i+1];
                float vjz = dvel[3*i+2];

                //collision
                float distsq = dx*dx + dy*dy + dz*dz;
                if (distsq < gtfo_distance * gtfo_distance){
                    float diff_factor = (gtfo_distance / distsq - rsqrtf(distsq)) * gtfo_distance;
                    avg_diffx -= diff_factor * dx;
                    avg_diffy -= diff_factor * dy;
                    avg_diffz -= diff_factor * dz;
                    collision_counter++;
                }
                if (distsq < alignment_distance * alignment_distance){
                    avg_velx += vjx;
                    avg_vely += vjy;
                    avg_velz += vjz;
                    alignment_counter++;
                }
                if (distsq < centering_distance * centering_distance){
                    avg_posx += jx;
                    avg_posy += jy;
                    avg_posz += jz;
                    centering_counter++;
                }
            }
            if (collision_counter > 0) {
                cox = avg_diffx / ((float)collision_counter);
                coy = avg_diffy / ((float)collision_counter);
                coz = avg_diffz / ((float)collision_counter);
            }
            alx = avg_velx / ((float)alignment_counter) - vx;
            aly = avg_vely / ((float)alignment_counter) - vy;
            alz = avg_velz / ((float)alignment_counter) - vz;
            cex = avg_posx / ((float)centering_counter) - x;
            cey = avg_posy / ((float)centering_counter) - y;
            cez = avg_posz / ((float)centering_counter) - z;

            float diff_top = fabs(10.0 - x);
            float diff_bottom = fabs(10.0 + x);
            float diff_right = fabs(10.0 - y);
            float diff_left = fabs(10.0 + y);
            float diff_front = fabs(10.0 - z);
            float diff_back = fabs(10.0 + z);
            if (diff_top < gtfo_distance) wax = (gtfo_distance / diff_top - 1) * gtfo_distance;
            if (diff_right < gtfo_distance) way = (gtfo_distance / diff_right - 1) * gtfo_distance;
            if (diff_front < gtfo_distance) waz = (gtfo_distance / diff_front - 1) * gtfo_distance;
            if (diff_bottom < gtfo_distance) wax = -(gtfo_distance / diff_bottom - 1) * gtfo_distance;
            if (diff_left < gtfo_distance) way = -(gtfo_distance / diff_left - 1) * gtfo_distance;
            if (diff_back < gtfo_distance) waz = -(gtfo_distance / diff_back - 1) * gtfo_distance;

            acx = (cox)*w_col + alx*w_ali + cex*w_cen - wax;
            acy = (coy)*w_col + aly*w_ali + cey*w_cen - way;
            acz = (coz)*w_col + alz*w_ali + cez*w_cen - waz;
            // printf("pos %i, %i, %f, %f, %f \n",
            //         index,
            //         time_step,
            //         x,
            //         y,
            //         z);
            // printf("vel %i, %i, %f, %f, %f \n",
            //         index,
            //         time_step,
            //         vx,
            //         vy,
            //         vz);
            // printf("acc %i, %i, %f, %f, %f \n",
            //         index,
            //         time_step,
            //         acx,
            //         acy,
            //         acz);
            // printf("co %i, %i, %f, %f, %f \n",
            //         index,
            //         time_step,
            //         cox,
            //         coy,
            //         coz);
            // printf("wa %i, %i, %f, %f, %f \n",
            //         index,
            //         time_step,
            //         wax,
            //         way,
            //         waz);
            // printf("al %i, %i, %f, %f, %f \n",
            //         index,
            //         time_step,
            //         alx,
            //         aly,
            //         alz);
            // printf("ce %i, %i, %f, %f, %f \n",
            //         index,
            //         time_step,
            //         cex,
            //         cey,
            //         cez);
        }
        __syncthreads();

        //physics update
        if (index < nboids){
            float speed = sqrtf(vx*vx + vy*vy + vz*vz);
            float dirx = vx / speed;
            float diry = vy / speed;
            float dirz = vz / speed;
            float acc_mag = sqrtf(acx*acx + acy*acy + acz*acz);
            // printf("%f, %f, %f, %f\n", acx, acy, acz, acc_mag);
            if (acc_mag > 0.00001){
                float adirx = acx / acc_mag;
                float adiry = acy / acc_mag;
                float adirz = acz / acc_mag;
                float dir_dots = dirx*adirx + diry*adiry + dirz*adirz;
                float acx_true = adirx - dirx * dir_dots;
                float acy_true = adiry - diry * dir_dots;
                float acz_true = adirz - dirz * dir_dots;
                float true_mag = sqrtf(acx_true*acx_true + acy_true*acy_true + acz_true*acz_true);
                acx_true = acx_true * acc_mag / true_mag;
                acy_true = acy_true * acc_mag / true_mag;
                acz_true = acz_true * acc_mag / true_mag;
                vx += acx_true * dt;
                vy += acy_true * dt;
                vz += acz_true * dt;
                speed = sqrtf(vx*vx + vy*vy + vz*vz);
                vx /= speed;
                vy /= speed;
                vz /= speed;
            }
            x += vx * dt;
            y += vy * dt;
            z += vz * dt;
            dpos[3*index] = x;
            dpos[3*index+1] = y;
            dpos[3*index+2] = z;
            dvel[3*index] = vx;
            dvel[3*index+1] = vy;
            dvel[3*index+2] = vz;

            //DATA_ARRAY ENTRY
            data_array[time_step*nboids*3 + 3*index] = x;
            data_array[time_step*nboids*3 + 3*index+1] = y;
            data_array[time_step*nboids*3 + 3*index+2] = z;
            // printf("%i, %i, %f, %f, %f \n",
            //         index,
            //         time_step,
            //         data_array[time_step*nboids*3 + 3*index],
            //         data_array[time_step*nboids*3 + 3*index+1],
            //         data_array[time_step*nboids*3 + 3*index+2]);
            // printf("\n");
        }
        __syncthreads();
    }

    // if (index < nboids)
    // for (int time_step = 0; time_step < steps; time_step++){
    //         printf("%i, %i, %f, %f, %f \n",
    //                 index,
    //                 time_step,
    //                 data_array[time_step*nboids*3 + 3*index],
    //                 data_array[time_step*nboids*3 + 3*index+1],
    //                 data_array[time_step*nboids*3 + 3*index+2]);
    // }
    
}

void boid::run(float time){
    steps = static_cast<int>(time / dt) + 1;
    int sim_boids_index = 0;
    sim_boids = new vec3[steps * nboids];

    for (int j = 0; j < nboids; j++){
        sim_boids[sim_boids_index] = pos[j];
        sim_boids_index++;
    }

    float *dpos = nullptr;
    float *dvel = nullptr;
    float *dacc = nullptr;
    float *ddim_low = nullptr;
    float *ddim_high = nullptr;
    float *dvel_low = nullptr;
    float *dvel_high = nullptr;
    float *data_array = nullptr;

    hipMalloc(&dpos, nboids*3*sizeof(float));
    hipMalloc(&dvel, nboids*3*sizeof(float));
    hipMalloc(&dacc, nboids*3*sizeof(float));
    hipMalloc(&ddim_low, 3*sizeof(float));
    hipMalloc(&ddim_high, 3*sizeof(float));
    hipMalloc(&dvel_low, 3*sizeof(float));
    hipMalloc(&dvel_high, 3*sizeof(float));
    hipMalloc(&data_array, 3*nboids*steps*sizeof(float));

    hipMemcpy(dpos, (float*)pos, nboids*3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dvel, (float*)vel, nboids*3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ddim_low, (float*)(&dim_low), 3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ddim_high, (float*)(&dim_high), 3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dvel_low, (float*)(&vel_low), 3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dvel_high, (float*)(&vel_high), 3*sizeof(float), hipMemcpyHostToDevice);

    sim_kernel<<<(nboids + 255)/256, 256>>>(
                nboids, steps, time, dt, gtfo_distance,
                centering_distance, alignment_distance,
                w_collision, w_alignment, w_centering,
                dpos, dvel, dacc,
                ddim_low, ddim_high,
                dvel_low, dvel_high,
                data_array);

    hipDeviceSynchronize();

    hipMemcpy((float*)(sim_boids), data_array, 3*nboids*steps*sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipFree(dpos);
    hipFree(dvel);
    hipFree(dacc);
    hipFree(ddim_high);
    hipFree(ddim_low);
    hipFree(dvel_high);
    hipFree(dvel_low);
    hipFree(data_array);
    
    // printf("bozo\n");
    // for (int time_step = 0; time_step < steps; time_step++){
    //         printf("%i, %i, %f, %f, %f \n",
    //                 0,
    //                 time_step,
    //                 ((float*)(sim_boids))[time_step*nboids*3 + 3*0],
    //                 ((float*)(sim_boids))[time_step*nboids*3 + 3*0+1],
    //                 ((float*)(sim_boids))[time_step*nboids*3 + 3*0+2]);
    // }

    write_sim_boids();
}

void boid::print_boids(){
    std::cout << std::fixed << std::setprecision(3);
    for (int i = 0; i < nboids; i++){
        std::cout << i << ": " << pos[i] << " + " << dt << "*" << vel[i] << std::endl;
    }
    std::cout << "center: " << center << std::endl;
}

void boid::write_sim_boids(){
    //write
    // int nboids
    // int steps
    // float dt
    // float time
    // float array 3*nboids*steps
    std::ofstream file("boid_data.bin", std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Failed to open the file." << std::endl;
        exit(-1);
    }
    file.write(reinterpret_cast<char*>(&nboids), sizeof(nboids));
    file.write(reinterpret_cast<char*>(&steps), sizeof(steps));
    file.write(reinterpret_cast<char*>(&dt), sizeof(dt));
    file.write(reinterpret_cast<char*>(&time), sizeof(time));
    file.write(reinterpret_cast<char*>(sim_boids), sizeof(vec3) * nboids * steps);
    file.close();
}

// calculate acceleration using all boids with each other
void boid::calc_acc_all(){
    vec3* collision = nullptr;
    vec3* alignment = nullptr;
    vec3* centering = nullptr;
    vec3* wall_avoidance_high = nullptr;
    vec3* wall_avoidance_low = nullptr;
    int counter = 0;

    // collision
    vec3 avg_diff = vec3(0,0,0);
    collision = new vec3[nboids];

    for (int i = 0; i < nboids; i++){
        counter = 0;
        avg_diff.clear();
        for (int j = 0; j < nboids; j++){
            if (j == i) continue;
            vec3 diff = (pos[i] - pos[j]);
            float distsq = diff.normsqrd();
            if (distsq < gtfo_distance * gtfo_distance){
                avg_diff += (diff * gtfo_distance / distsq - diff / diff.norm()) * gtfo_distance;
                counter++;
            }
        }
        if (counter > 0) {
            avg_diff /= counter;
            collision[i] = avg_diff;
        }
    }

    // alignment
    vec3* avg_vel = new vec3[nboids];
    alignment = new vec3[nboids];

    for (int i = 0; i < nboids; i++){
        int alignment_counter = 1;
        avg_vel[i] = vec3(0,0,0);
        for (int j = 0; j < nboids; j++){
            if ((pos[i] - pos[j]).norm() <= alignment_distance){
                alignment_counter++;
                avg_vel[i] = avg_vel[i] + vel[j];
            }
        }
        avg_vel[i] = avg_vel[i] / ((float)alignment_counter);
    }

    for (int i = 0; i < nboids; i++){
        alignment[i] = avg_vel[i] - vel[i];
    }

    // centering
    vec3* avg_pos = new vec3[nboids];
    centering = new vec3[nboids];

    for (int i = 0; i < nboids; i++){
        int centering_counter = 1;
        avg_pos[i] = vec3(0,0,0);
        for (int j = 0; j < nboids; j++){
            if ((pos[i] - pos[j]).norm() <= centering_distance){
                centering_counter++;
                avg_pos[i] = avg_pos[i] + pos[j];
            }
        }
        avg_pos[i] = avg_pos[i] / ((float)centering_counter);
    }

    for (int i = 0; i < nboids; i++){
        centering[i] = avg_pos[i] - pos[i];
    }

    // walls
    wall_avoidance_high = new vec3[nboids];
    wall_avoidance_low = new vec3[nboids];

    for (int i = 0; i < nboids; i++){
        float diff_top = std::fabs(pos[i].x - dim_high.x);
        float diff_bottom = std::fabs(pos[i].x - dim_low.x);

        float diff_right = std::fabs(pos[i].y - dim_high.y);
        float diff_left = std::fabs(pos[i].y - dim_low.y);

        float diff_front = std::fabs(pos[i].z - dim_high.z);
        float diff_back = std::fabs(pos[i].z - dim_low.z);

        if (diff_top < gtfo_distance){
            wall_avoidance_high[i].x = -(gtfo_distance / diff_top - 1) * gtfo_distance;
        }
        
        if (diff_right < gtfo_distance){
            wall_avoidance_high[i].y = -(gtfo_distance / diff_right - 1) * gtfo_distance;
        }

        if (diff_front < gtfo_distance){
            wall_avoidance_high[i].z = -(gtfo_distance / diff_front - 1) * gtfo_distance;
        }

        if (diff_bottom < gtfo_distance){
            wall_avoidance_low[i].x = (gtfo_distance / diff_bottom - 1) * gtfo_distance;
        }

        if (diff_left < gtfo_distance){
            wall_avoidance_low[i].y = (gtfo_distance / diff_left - 1) * gtfo_distance;
        }

        if (diff_back < gtfo_distance){
            wall_avoidance_low[i].z = (gtfo_distance / diff_back - 1) * gtfo_distance;
        }
    }


    // calculate acceleration for all boids
    for (int i = 0; i < nboids; i++){
        acc[i] = (collision[i] * w_collision) + (alignment[i] * w_alignment) + (centering[i] * w_centering) + wall_avoidance_high[i] + wall_avoidance_low[i];
    }

    delete [] alignment;
    delete [] collision;
    delete [] centering;
}

void boid::set_center_all(){
    center.clear();
    for (int i = 0; i < nboids; i++){
        center += pos[i];
    }
    center /= nboids;
}

void boid::physics_update(){
    for(int i = 0; i < nboids; i++){
        vec3 dir = vel[i].normalized();
        vec3 acc_dir = acc[i].normalized();
        float dot_product = dir.x * acc_dir.x + dir.y * acc_dir.y + dir.z * acc_dir.z;
        vec3 true_acc_dir = (acc_dir - dir * dot_product).normalized();
        float true_acc_mag = acc[i].norm();
        // if (true_acc_mag >= 3) true_acc_mag = 3;
        vel[i] += true_acc_dir * true_acc_mag * dt;
        vel[i].normalize();
        pos[i] += vel[i] * dt;
        // vel[i] += acc[i] * dt;
        // if (vel[i].normsqrd() >= 1){
        //     vel[i] /= vel[i].norm();
        // }
        // pos[i] += vel[i] * dt;
    }
}
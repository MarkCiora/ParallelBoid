#include "hip/hip_runtime.h"
#include "boid.h"

#include <algorithm>
#include <random>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>
#include <fstream>
#include <cmath>

#include "vec3.h"

vec3* boid::pos = nullptr;
vec3* boid::vel = nullptr;
vec3* boid::acc = nullptr;
vec3* boid::sim_boids = nullptr;
vec3 boid::dim_low = vec3(-10., -10., -10.);
vec3 boid::dim_high = vec3(10., 10., 10.);
vec3 boid::vel_low = vec3(-1., -1., -1.);
vec3 boid::vel_high = vec3(1., 1., 1.);
vec3 boid::center = vec3(0., 0., 0.);
int boid::nboids = 2;
int boid::steps = 0;
float boid::dt = 1. / (float)(60);
float boid::time = 0.0;
float boid::centering_distance = 1.5;
float boid::alignment_distance = 1.;

float gtfo_distance = 1;
float boid::w_collision = 0.4;
float boid::w_alignment = 0.4;
float boid::w_centering = 0.3;

void boid::new_boids_random(){
    kill();
    pos = new vec3[nboids];
    vel = new vec3[nboids];
    acc = new vec3[nboids];
    vec3 dim_diff = dim_high - dim_low;
    vec3 vel_diff = vel_high - vel_low;
    for (int i = 0; i < 3*nboids; i++){
        ((float*)pos)[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        ((float*)pos)[i] *= ((float*)&dim_diff)[i%3];
        ((float*)pos)[i] += ((float*)&dim_low)[i%3];
        ((float*)vel)[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        ((float*)vel)[i] *= ((float*)&vel_diff)[i%3];
        ((float*)vel)[i] += ((float*)&vel_low)[i%3];
        ((float*)acc)[i] = 0;
    }
    for (int i = 0; i < nboids; i++) vel[i].normalize();
}

void boid::kill(){
    delete [] pos;
    delete [] vel;
    delete [] acc;
    delete [] sim_boids;
}

void boid::step_sim(){
    calc_acc_all();
    physics_update();
    time += dt;
}

__global__
void sim_kernel(int nboids, int steps, float time, float dt,
                float centering_distance, float alignment_distance,
                float w_col, float w_ali, float w_cen,
                float* dpos, float *dvel, float *dacc,
                float* ddim_low, float* ddim_high,
                float* dvel_low, float* dvelhigh,
                float* dcenter){

}

void boid::run(float time){
    steps = static_cast<int>(time / dt) + 1;
    int sim_boids_index = 0;
    sim_boids = new vec3[steps * nboids];

    for (int j = 0; j < nboids; j++){
        sim_boids[sim_boids_index] = pos[j];
        sim_boids_index++;
    }

    float *dpos = nullptr;
    float *dvel = nullptr;
    float *dacc = nullptr;
    float *ddim_low = nullptr;
    float *ddim_high = nullptr;
    float *dvel_low = nullptr;
    float *dvel_high = nullptr;
    float *dcenter = nullptr;

    sim_kernel<<<1,1>>>( nboids, steps, time, dt,
                centering_distance, alignment_distance,
                w_collision, w_alignment, w_centering,
                dpos, dvel, dacc,
                ddim_low, ddim_high,
                dvel_low, dvel_high,
                dcenter);

    // for (int i = 1; i < steps; i++){
    //     step_sim();
    //     for (int j = 0; j < nboids; j++){
    //         sim_boids[sim_boids_index] = pos[j];
    //         sim_boids_index++;
    //     }
    // }

    write_sim_boids();
}

void boid::print_boids(){
    std::cout << std::fixed << std::setprecision(3);
    for (int i = 0; i < nboids; i++){
        std::cout << i << ": " << pos[i] << " + " << dt << "*" << vel[i] << std::endl;
    }
    std::cout << "center: " << center << std::endl;
}

void boid::write_sim_boids(){
    //write
    // int nboids
    // int steps
    // float dt
    // float time
    // float array 3*nboids*steps
    std::ofstream file("boid_data.bin", std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Failed to open the file." << std::endl;
        exit(-1);
    }
    file.write(reinterpret_cast<char*>(&nboids), sizeof(nboids));
    file.write(reinterpret_cast<char*>(&steps), sizeof(steps));
    file.write(reinterpret_cast<char*>(&dt), sizeof(dt));
    file.write(reinterpret_cast<char*>(&time), sizeof(time));
    file.write(reinterpret_cast<char*>(sim_boids), sizeof(vec3) * nboids * steps);
    file.close();
}

// calculate acceleration using all boids with each other
void boid::calc_acc_all(){
    vec3* collision = nullptr;
    vec3* alignment = nullptr;
    vec3* centering = nullptr;
    vec3* wall_avoidance_high = nullptr;
    vec3* wall_avoidance_low = nullptr;
    int counter = 0;

    // collision
    vec3 avg_diff = vec3(0,0,0);
    collision = new vec3[nboids];

    for (int i = 0; i < nboids; i++){
        counter = 0;
        avg_diff.clear();
        for (int j = 0; j < nboids; j++){
            if (j == i) continue;
            vec3 diff = (pos[i] - pos[j]);
            float distsq = diff.normsqrd();
            if (distsq < gtfo_distance * gtfo_distance){
                avg_diff += (diff * gtfo_distance / distsq - diff / diff.norm()) * gtfo_distance;
                counter++;
            }
        }
        if (counter > 0) {
            avg_diff /= counter;
            collision[i] = avg_diff;
        }
    }

    // alignment
    vec3* avg_vel = new vec3[nboids];
    alignment = new vec3[nboids];

    for (int i = 0; i < nboids; i++){
        int alignment_counter = 1;
        avg_vel[i] = vec3(0,0,0);
        for (int j = 0; j < nboids; j++){
            if ((pos[i] - pos[j]).norm() <= alignment_distance){
                alignment_counter++;
                avg_vel[i] = avg_vel[i] + vel[j];
            }
        }
        avg_vel[i] = avg_vel[i] / ((float)alignment_counter);
    }

    for (int i = 0; i < nboids; i++){
        alignment[i] = avg_vel[i] - vel[i];
    }

    // centering
    vec3* avg_pos = new vec3[nboids];
    centering = new vec3[nboids];

    for (int i = 0; i < nboids; i++){
        int centering_counter = 1;
        avg_pos[i] = vec3(0,0,0);
        for (int j = 0; j < nboids; j++){
            if ((pos[i] - pos[j]).norm() <= centering_distance){
                centering_counter++;
                avg_pos[i] = avg_pos[i] + pos[j];
            }
        }
        avg_pos[i] = avg_pos[i] / ((float)centering_counter);
    }

    for (int i = 0; i < nboids; i++){
        centering[i] = avg_pos[i] - pos[i];
    }

    // walls
    wall_avoidance_high = new vec3[nboids];
    wall_avoidance_low = new vec3[nboids];

    for (int i = 0; i < nboids; i++){
        float diff_top = std::fabs(pos[i].x - dim_high.x);
        float diff_bottom = std::fabs(pos[i].x - dim_low.x);

        float diff_right = std::fabs(pos[i].y - dim_high.y);
        float diff_left = std::fabs(pos[i].y - dim_low.y);

        float diff_front = std::fabs(pos[i].z - dim_high.z);
        float diff_back = std::fabs(pos[i].z - dim_low.z);

        if (diff_top < gtfo_distance){
            wall_avoidance_high[i].x = -(gtfo_distance / diff_top - 1) * gtfo_distance;
        }
        
        if (diff_right < gtfo_distance){
            wall_avoidance_high[i].y = -(gtfo_distance / diff_right - 1) * gtfo_distance;
        }

        if (diff_front < gtfo_distance){
            wall_avoidance_high[i].z = -(gtfo_distance / diff_front - 1) * gtfo_distance;
        }

        if (diff_bottom < gtfo_distance){
            wall_avoidance_low[i].x = (gtfo_distance / diff_bottom - 1) * gtfo_distance;
        }

        if (diff_left < gtfo_distance){
            wall_avoidance_low[i].y = (gtfo_distance / diff_left - 1) * gtfo_distance;
        }

        if (diff_back < gtfo_distance){
            wall_avoidance_low[i].z = (gtfo_distance / diff_back - 1) * gtfo_distance;
        }
    }


    // calculate acceleration for all boids
    for (int i = 0; i < nboids; i++){
        acc[i] = (collision[i] * w_collision) + (alignment[i] * w_alignment) + (centering[i] * w_centering) + wall_avoidance_high[i] + wall_avoidance_low[i];
    }

    delete [] alignment;
    delete [] collision;
    delete [] centering;
}

void boid::set_center_all(){
    center.clear();
    for (int i = 0; i < nboids; i++){
        center += pos[i];
    }
    center /= nboids;
}

void boid::physics_update(){
    for(int i = 0; i < nboids; i++){
        vec3 dir = vel[i].normalized();
        vec3 acc_dir = acc[i].normalized();
        float dot_product = dir.x * acc_dir.x + dir.y * acc_dir.y + dir.z * acc_dir.z;
        vec3 true_acc_dir = (acc_dir - dir * dot_product).normalized();
        float true_acc_mag = acc[i].norm();
        // if (true_acc_mag >= 3) true_acc_mag = 3;
        vel[i] += true_acc_dir * true_acc_mag * dt;
        vel[i].normalize();
        pos[i] += vel[i] * dt;
        // vel[i] += acc[i] * dt;
        // if (vel[i].normsqrd() >= 1){
        //     vel[i] /= vel[i].norm();
        // }
        // pos[i] += vel[i] * dt;
    }
}